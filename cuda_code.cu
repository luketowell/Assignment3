#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <omp.h>
#include <stdlib.h>

// Code for the parallel CUDA code
__global__ void exponentialFunction (int dataPoints, float *devX, float *devY)
{
   float first, second, third;
   int my_i = blockIdx.x*blockDim.x + threadIdx.x;
	
   if (my_i<dataPoints+1){
      first = ((devX[my_i]-2))*((devX[my_i]-2));
      second = (pow((devX[my_i]-6.0),2)/10);
      third = (1/(pow((double)devX[my_i],2.0)+1)); 
      devY[my_i] = (expf(-first)+expf(-second)+third);
   }
	
}

__global__ void initX (float *devX, int dataPoints, float discretePoint, int threads)
{
    int my_i = blockIdx.x*blockDim.x + threadIdx.x;
 
    if (my_i < dataPoints+1){
       devX[my_i] = (discretePoint * my_i)-100; 
    }
}

// Serial Code
void serialFunction (int dataPoints, float *serialX, float *serialY)
{
   float first, second, third;
   int i;

   for(i=0; i < dataPoints+1; i++)
      {
         first = ((serialX[i]-2))*((serialX[i]-2));
         second = (pow((serialX[i]-6),2)/10);
         third = (1/(pow(serialX[i],2)+1));
         serialY[i] = (exp(-first)+exp(-second)+third);
      }

   printf("ran serial code \n");
}

int main(int argc, char **argv)
{
   int i, numGPU;
   	
   hipGetDeviceCount(&numGPU);
   if (numGPU >= 1) {
      // Steps between the two outer values e.g. -100 and +100.
      float steps = 200;

      // set the amoutn of dataPoints that you want to discretize the function over
      int dataPoints = strtol(argv[1], NULL, 10);

      //set number of omp threads to be used;
      omp_set_num_threads(12);
      int ompthreads = omp_get_max_threads();
      printf("number of omp threads = %d", ompthreads);
 
      // X and F(x) as Y declaration
      float *X, *Y, *serialX, *serialY, *cudaX;
      float *devX, *devY, *devCudaX;
      float maxY, serialMaxY;
  
      //create cuda timing objects
      hipEvent_t cudaIStart, cudaIEnd, startCuda, stopCuda;
      hipEventCreate(&cudaIStart);
      hipEventCreate(&cudaIEnd);
      hipEventCreate(&startCuda);
      hipEventCreate(&stopCuda);
      
      //declare and work out the number of threads and blocks.
      int potentialBlocks, maxBlocks = 65535; 
      int threads = strtol(argv[2], NULL, 10);

      // Ternary statement for working out amout of blocks to be used.
      potentialBlocks = ceil((float)dataPoints/(float)threads);
      int blocks = (potentialBlocks<maxBlocks) ? potentialBlocks : maxBlocks ;

      printf("using %d threads on %d blocks \n", threads, blocks);

      //OMP timing variables
      double cudaStart, cudaInitEnd,cudaFuncMemStart, cudaFuncMemEnd, cudaEnd;
      double serialFunctionStart, serialFunctionEnd, serialStart, serialEnd, serialInitStart, serialInitEnd, serialMaxStart, serialMaxEnd;
      double ompMaxStart, ompMaxEnd;
      
      // Device memory allocation
      hipMalloc(&devX, dataPoints*sizeof(float));
      hipMalloc(&devY, dataPoints*sizeof(float));
      hipMalloc(&devCudaX, dataPoints*sizeof(float));

      //Host Memory Allocation
      cudaX = (float *) malloc(sizeof(float)*dataPoints);
      X = (float *) malloc(sizeof(float)*dataPoints);
      Y = (float *) malloc(sizeof(float)*dataPoints);
      serialX = (float *) malloc(sizeof(float)*dataPoints);
      serialY = (float *) malloc(sizeof(float)*dataPoints);
      
      //Start executing
      cudaStart = omp_get_wtime();	
      float discretePoint = steps/dataPoints;

      //init the values of X cuda timings
      hipEventRecord(cudaIStart,0);

      initX<<<blocks, threads>>>(devX, dataPoints, discretePoint, threads);
      
      hipEventRecord(cudaIEnd);
      hipEventSynchronize(cudaIEnd);
      hipMemcpy(X, devX, dataPoints*sizeof(float), hipMemcpyDeviceToHost);
      cudaInitEnd = omp_get_wtime();	

      //cuda initialisation timing
      float iTime;
      hipEventElapsedTime(&iTime, cudaIStart, cudaIEnd);
    
      cudaFuncMemStart = omp_get_wtime();
      // Copy the host contents of X over to device devX
      hipMemcpy(devX, X, dataPoints*sizeof(float), hipMemcpyHostToDevice);	
    
      // Check for errors after Copying X over to new Device
      hipError_t err = hipGetLastError();
      if (err != hipSuccess) {
        printf("(1) CUDA RT error: %s \n", hipGetErrorString(err));
      }

      //Start the Cuda Timings
      hipEventRecord(startCuda, 0);

      //Call the function kernel
      exponentialFunction<<<blocks,threads>>> (dataPoints, devX, devY);
      //Stop the Cuda Timings
      hipEventRecord(stopCuda);
      hipEventSynchronize(stopCuda);
      // check for errors after running Kernel
      err = hipGetLastError();
      if (err != hipSuccess) {
        printf("(2) CUDA RT error: %s \n", hipGetErrorString(err));
      }

      // Copy over the Y value from the device to the host
      hipMemcpy(Y, devY, dataPoints*sizeof(float), hipMemcpyDeviceToHost);
      cudaFuncMemEnd=omp_get_wtime();
      //Check for errors after copying errors over from device to host.
      err = hipGetLastError();
      if (err != hipSuccess) {
        printf("(3) CUDA RT error: %s \n", hipGetErrorString(err));
      }

      //clean up memory
      hipFree(devX);
      hipFree(devY);
	
      //Work out time
      float cTime;
      hipEventElapsedTime(&cTime, startCuda, stopCuda);
      
      ompMaxStart = omp_get_wtime();
      //print out the Cuda+OMP result
      #pragma omp parallel for default(none) shared(Y, dataPoints) private(i) reduction(max: maxY) 
      for(i=0; i < dataPoints+1; i++)
      {
         if (Y[i] > maxY){
             maxY = Y[i];
         }
      }
      ompMaxEnd = omp_get_wtime();
      //end of cuda+omp implementation
      cudaEnd = omp_get_wtime();

      //start serial timings
      serialStart = omp_get_wtime();
      
      //work out discrete point again for serial
      float serialDiscretePoint = steps/dataPoints;


      //discretise the range to work out X[i]
      serialInitStart = omp_get_wtime();

      for (i= 0; i < dataPoints+1; i++){
        serialX[i] = (serialDiscretePoint * i)-100;
      }
      
      serialInitEnd = omp_get_wtime();

      //call the serial code:
      serialFunctionStart = omp_get_wtime(); 
      serialFunction(dataPoints,serialX, serialY);
      serialFunctionEnd = omp_get_wtime();
      
      serialMaxStart = omp_get_wtime();
      //work out max in serial
      for(i=0; i < dataPoints+1; i++)
      {
         if (serialY[i] > serialMaxY){
             serialMaxY = Y[i];
         }
      }
      serialMaxEnd = omp_get_wtime();

      //end serial timings
      serialEnd = omp_get_wtime();
      
      //total timings
      printf("cuda init kernel with memory transfer: %0.5f\n", (cudaInitEnd - cudaStart)*1000);
      printf("cuda init kernel : %0.8f\n", iTime);
      printf("omp init %0.5f\n", (ompInitEnd - ompInitStart)*1000);
      printf("cuda function with memory transfer: %0.5f\n", (cudaFuncMemEnd - cudaFuncMemStart)*1000);
      printf("cuda function: %0.5f\n", cTime);
      printf("omp max calc: %0.5f\n", (ompMaxEnd - ompMaxStart)*1000);
      printf("total cuda Time: %0.5f\n", (cudaEnd - cudaStart)*1000);
      printf("serial init %0.5f\n", (serialInitEnd - serialInitStart)*1000);
      printf("serial function: %0.5f\n", (serialFunctionEnd-serialFunctionStart)*1000);
      printf("serial max calc: %0.5f\n", (serialMaxEnd - serialMaxStart)*1000);
      printf("all serial: %0.5f \n", (serialEnd - serialStart) * 1000);
      printf("cuda+omp maxY: %0.8f\n", maxY);
      printf("serial maxY: %0.8f\n", serialMaxY);

   }
   else
   {
    printf("No GPUs are detected!\n");
   }
}