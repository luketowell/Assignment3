
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <omp.h>
#include <stdlib.h>

// Code for the parallel CUDA code
__global__ void exponentialFunction (int dataPoints, float *devX, float *devY)
{
   float first, second, third;
   int my_i = blockIdx.x*blockDim.x + threadIdx.x;
	
   if (my_i<dataPoints+1){
      first = ((devX[my_i]-2))*((devX[my_i]-2));
      second = (pow((devX[my_i]-6.0),2)/10);
      third = (1/(pow((double)devX[my_i],2.0)+1)); 
      devY[my_i] = (expf(-first)+expf(-second)+third);
   }
	
}

// Serial Code
void serialFunction (int dataPoints, float *serialX, float *serialY)
{
   float first, second, third;
   int i;

   for(i=0; i < dataPoints+1; i++)
      {
         first = ((serialX[i]-2))*((serialX[i]-2));
         second = (pow((serialX[i]-6),2)/10);
         third = (1/(pow(serialX[i],2)+1));
         serialY[i] = (exp(-first)+exp(-second)+third);
      }

   printf("ran serial code \n");
}

int main(int argc, char **argv)
{
   int i, numGPU;
	
   hipGetDeviceCount(&numGPU);
   if (numGPU >= 1) {

      float steps = 200;
      int dataPoints = strtol(argv[1], NULL, 10);
     
      // X and F(x) as Y declaration
      float *X, *Y, *serialX, *serialY;
      float *devX, *devY;
      float maxY, serialMaxY;
  
      //create cuda timing objects
      hipEvent_t startCuda, stopCuda;
      hipEventCreate(&startCuda);
      hipEventCreate(&stopCuda);

      //OMP timing variables
      double cudaStart, cudaEnd, serialFunctionStart, serialFunctionEnd, serialStart, serialEnd, serialInitStart, serialInitEnd, ompInitStart, ompInitEnd;
      
      // Device memory allocation
      hipMalloc(&devX, dataPoints*sizeof(float));
      hipMalloc(&devY, dataPoints*sizeof(float));

      //Host Memory Allocation
      X = (float *) malloc(sizeof(float)*dataPoints);
      Y = (float *) malloc(sizeof(float)*dataPoints);
      serialX = (float *) malloc(sizeof(float)*dataPoints);
      serialY = (float *) malloc(sizeof(float)*dataPoints);
      
      //Start executing
      cudaStart = omp_get_wtime();	
      float discretePoint = steps/dataPoints;


      //discretise the range to work out X[i]
      ompInitStart = omp_get_wtime();
      #pragma omp parallel for default(none) shared(dataPoints, X, discretePoint) private(i)
      for (i= 0; i < dataPoints+1; i++){
         X[i] = (discretePoint * i)-100;
      }
      ompInitEnd = omp_get_wtime();

      // Copy the host contents of X over to device devX
      hipMemcpy(devX, X, dataPoints*sizeof(float), hipMemcpyHostToDevice);	

      // Check for errors after Copying X over to new Device
      hipError_t err = hipGetLastError();
      if (err != hipSuccess) {
        printf("(1) CUDA RT error: %s \n", hipGetErrorString(err));
      }
  
      //Work out threads and blocks and print out number of threads and blocks
      int threads = strtol(argv[2], NULL, 10);
      int blocks = ceil((float)dataPoints/(float)threads);
      printf("using %d threads on %d blocks \n", threads, blocks);


      //Start the Cuda Timings
      hipEventRecord(startCuda, 0);

      //Call the function kernel
      exponentialFunction<<<blocks,threads>>> (dataPoints, devX, devY);
      //Stop the Cuda Timings
      hipEventRecord(stopCuda, 0);
   
      // check for errors after running Kernel
      err = hipGetLastError();
      if (err != hipSuccess) {
        printf("(2) CUDA RT error: %s \n", hipGetErrorString(err));
      }

      // Copy over the Y value from the device to the host
      hipMemcpy(Y, devY, dataPoints*sizeof(float), hipMemcpyDeviceToHost);
   
      //Check for errors after copying errors over from device to host.
      err = hipGetLastError();
      if (err != hipSuccess) {
        printf("(3) CUDA RT error: %s \n", hipGetErrorString(err));
      }

      //clean up memory
      hipFree(devX);
      hipFree(devY);
	
      cudaEnd = omp_get_wtime();
      //Work out time
      float cTime;
      hipEventElapsedTime(&cTime, startCuda, stopCuda);
      
 
      //print out the Cuda+OMP result and timing
      for(i=0; i < dataPoints+1; i++)
      {
         printf("X = %0.5f \n", X[i]);
         printf("Y = %0.5f \n", Y[i]);
      }

      //start serial timings
      serialStart = omp_get_wtime();
      
      //work out discrete point again for serial
      discretePoint = steps/dataPoints;


      //discretise the range to work out X[i]
      serialInitStart = omp_get_wtime();

      for (i= 0; i < dataPoints+1; i++){
         X[i] = (discretePoint * i)-100;
      }
      
      serialInitEnd = omp_get_wtime();

      //call the serial code:
      serialFunctionStart = omp_get_wtime(); 
      serialFunction(dataPoints, X, Y);
      serialFunctionEnd = omp_get_wtime();

      //work out max in serial

      //end serial timings
      serialEnd = omp_get_wtime();
      
      //total timings
      printf("omp init %0.5f\n", (ompInitEnd - ompInitStart)*1000);
      printf("cuda function: %0.5f\n", cTime);
      printf("total cuda Time: %0.5f\n", (cudaEnd - cudaStart)*1000);
      printf("serial init %0.5f\n", (serialInitEnd - serialInitStart)*1000);
      printf("serial function: %0.5f\n", (serialFunctionEnd-serialFunctionStart)*1000);
      printf("all serial: %0.5f \n", (serialEnd - serialStart) * 1000);

   }
   else
   {
    printf("No GPUs are detected!\n");
   }
}